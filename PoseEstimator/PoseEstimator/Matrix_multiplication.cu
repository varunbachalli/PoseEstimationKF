#include "hip/hip_runtime.h"
#include "Matrix_multiplication.cuh"
#include <fstream>
#include <string>
#include <chrono>

__global__ void multiply(double* A, double* A_t, int numvalues)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < numvalues)
	{
		*(A_t + id) = (*(A + id)) * (*(A_t + id));
	}
}


__global__ void ATA(double* A, double* A_t, double* C, int num_values_in_A) // A and A_t = numstates x numsamples 
{
	int id = blockIdx.x* blockDim.x + threadIdx.x;
	int num_samples = num_values_in_A / blockDim.x;
	if (id == 0)
	{
		multiply <<<num_values_in_A/512 + 1, 512>>>(A, A_t,num_values_in_A);
	}

	hipDeviceSynchronize();
	__syncthreads();
	for (int i = 0; i < num_samples; ++i)
	{
		*(C + id) += *(A_t + id * num_samples + i);
	}
}

__global__ void ATb(double* A, double* b, int num_samples)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	b[id] = 0;
	for (int i = 0; i < num_samples; ++i)
	{
		b[id] += A[id * num_samples + i];
	}
}

int numThreads(int numstates)
{
	int k = 0;
	if ((numstates + 1) % 2 == 1)
	{
		k = (numstates + 1) / 2 + 1;
	}
	else
	{
		k = (numstates + 1) / 2;
	}

	return k;
}

double* PermutateA(double* A, int num_samples, int num_states, int num_shifts)
{
	double* A_perm = (double*)malloc(sizeof(double) * num_samples * num_states);

	int size_block1 = sizeof(double) * (num_samples) * (num_states - num_shifts); // chunk that goes to the front 
	int size_block2 = sizeof(double) * (num_samples) * (num_shifts); // chunk that goes to the back

	memcpy(A_perm, (A + (num_samples) * (num_shifts)), size_block1);
	memcpy(A_perm + (num_samples) * (num_states - num_shifts), A, size_block2);
	return A_perm;
}

void printValues(double* A, int numstates, int numsamples)
{
	for (int i = 0; i < numstates; ++i)
	{
		for (int j = 0; j < numsamples; ++j)
		{
			std::cout << *(A + i * numsamples + j) << " , ";

		}
		std::cout << std::endl;
	}
}


LeastSquares MatrixTransposeMultiplication(double* A, int numstates, int numsamples)
{
	int nStreams = numThreads(numstates);
	std::cout << "these many streams need to be made" << std::endl;
	std::cout << nStreams << std::endl;

	double* C = (double*)malloc(sizeof(double) * nStreams * numstates);
	double* b = (double*)malloc(sizeof(double) * numstates);
	std::vector<double*> pointersToA_perm_matrices;
	hipStream_t* cudaStreams = new hipStream_t[nStreams+1]; // nStreams for ATA, 1 stream for ATb
	
	for (int i = 0; i < nStreams; ++i) // create n streams
	{
		hipStreamCreate(&cudaStreams[i]);
		pointersToA_perm_matrices.push_back(PermutateA(A, numsamples, numstates, i));
	}

	hipStreamCreate(&cudaStreams[nStreams]); // create stream for ATb

	double* d_A;
	double* d_A_transpose;
	double* dC;
	double* db;
	hipMalloc(&d_A, sizeof(double) * numstates * numsamples);
	hipMalloc(&d_A_transpose, sizeof(double) * numstates * numsamples * nStreams);
	hipMalloc(&dC, sizeof(double) * nStreams * numstates);
	hipMalloc(&db, sizeof(double) * numstates);
	hipMemcpy(d_A, A, sizeof(double) * numstates * numsamples,hipMemcpyHostToDevice);
	
	int streamSize = numstates * numsamples;

	for (int i = 0; i < nStreams; ++i)
	{
		int offset = i * streamSize;
		hipMemcpyAsync(d_A_transpose + offset, pointersToA_perm_matrices[i], sizeof(double) * numstates * numsamples, hipMemcpyHostToDevice, cudaStreams[i]);
	}

	for (int i = 0; i < nStreams; ++i) {
		int offset = i * streamSize;
		ATA<<<1, numstates, 0, cudaStreams[i]>>> (d_A, d_A_transpose + offset, dC + i*numstates, numsamples*numstates);
	}

	ATb <<<1, numstates, 0, cudaStreams[nStreams] >>> (d_A, db, numsamples); // ATb

	for (int i = 0; i < nStreams; ++i) {
		hipMemcpyAsync(C + i * numstates, dC + i * numstates, sizeof(double)*numstates, hipMemcpyDeviceToHost, cudaStreams[i]);
	}
	hipMemcpyAsync(b, db, sizeof(double) * numstates, hipMemcpyDeviceToHost, cudaStreams[nStreams]);


	for (int i = 0; i < nStreams+1; ++i) {
		hipStreamSynchronize(cudaStreams[i]);
	}
	
	/*std::cout << "above is c Printed " << std::endl;
	printValues((double*)C, nStreams, numstates);*/

	

	for (int i = 0; i < nStreams+1; ++i) 
	{
		hipStreamDestroy(cudaStreams[i]);
	}
	
	LeastSquares Solution;
	Solution.ATA = new double[numstates * numstates];
	Solution.ATb = b;
	
	for (int i = 0; i < nStreams; ++i)
	{
		for (int k = 0; k < numstates; ++k)
		{
			if ((i + k) < numstates)
			{
				Solution.ATA[k * numstates + (i + k)] = C[numstates * i + k];
				Solution.ATA[(i + k) * numstates + k] = C[numstates * i + k];
			}
			else
			{
				Solution.ATA[numstates * (i + k - numstates) + k] = C[numstates * i + k];
				Solution.ATA[k * numstates + (i + k - numstates)] = C[numstates * i + k];
			}
		}
	}

	Solution.numstates = numstates;
	delete [] cudaStreams;
	hipFree(d_A);
	hipFree(d_A_transpose);
	hipFree(dC);
	hipFree(db);
	free(C);
	return Solution;
}

double* getA()
{
	double* A = (double*)malloc(6000 * sizeof(double));
	std::ifstream file;
	file.open("D:/GITProjects/Kalman Filtering Server/PoseEstimationKF/Sensor_CSV/myfile.txt");
	if (!file)
	{
		std::cerr << "couldn't open file" << std::endl;
	}
	else
	{
		for (int i = 0; i < 6000; ++i)
		{
			std::string outstring;
			file >> outstring;
			*(A + i) = std::stod(outstring);
		}
	}
	std::cout << "could read the file \n";

	return A;
}


//
//int main()
//{
//	double* A_ = getA();
//	int numberofDataPoints = 1000;
//	int numberofStates = 6;
//
//	//double A_[] = { 5, 7, 9, 9, 9, 0, 3, 5, 5, 7,
//	//				2, 7, 7, 2, 8, 4, 7, 0, 5, 0,
//	//				8, 4, 2, 3, 3, 9, 0, 5, 1, 6,
//	//				8, 2, 3, 2, 0, 7, 8, 8, 3, 8,
//	//				5, 5, 8, 9, 2, 4, 8, 3, 5, 5,
//	//				0, 4, 7, 4, 3, 5, 4, 8, 6, 3 };
//
//
//
//
//	LeastSquares Solution;
//
//	/*
//	Solution.ATA = Eigen::Map<Eigen::MatrixXd>(A_, numberofDataPoints, numberofStates);
//	Solution.ATb = Eigen::Map<Eigen::VectorXd>(A_, numberofDataPoints* numberofStates);
//	std::cout << "first" << std::endl;
//	std::cout << Solution.ATA << std::endl;
//	std::cout << "second" << std::endl;
//	std::cout << Solution.ATb << std::endl;
//	*/
//
//
//	/*printf("original matrix is");
//	printValues((double*)A_, numberofStates, numberofDataPoints);*/
//	auto start = std::chrono::high_resolution_clock::now();
//	Solution = MatrixTransposeMultiplication((double*)A_, numberofStates, numberofDataPoints);
//	std::cout << "Solution.ATA is" << std::endl;
//	printValues(Solution.ATA, numberofStates, numberofStates);
//
//	std::cout << "b is " << std::endl;
//	std::cout << "[";
//	for (int i = 0; i < numberofStates; ++i)
//	{
//		std::cout << Solution.ATb[i] << "\t";
//	}
//	std::cout << "]\n";
//
//	free(Solution.ATA);
//	free(Solution.ATb);
//	free(A_);
//
//	auto stop = std::chrono::high_resolution_clock::now();
//
//	auto duration = std::chrono::duration_cast <std::chrono::milliseconds> (stop - start);
//	std::cout << "time taken is " << duration.count() << std::endl;
//	getch();
//
//	return 0;
//
//	//double Answer[] = { 425 ,258 ,212 ,234 ,330, 257,
//	//					258 ,260 ,129 ,154 ,232, 187,
//	//					212 ,129 ,245 ,238 ,195, 160,
//	//					234 ,154 ,238 ,331 ,263, 210,
//	//					330 ,232 ,195 ,263 ,338, 239,
//	//					257 ,187 ,160 ,210 ,239, 240 };
//
//	//
//
//	//std::cout << "answer is supposed to be" << std::endl;
//	//printValues((double*)Answer, 6, 6);
//
//	//double b[] = { 59,42,41,49,54,44.0 };
//	//std::cout << "b is supposed to be " << std::endl;
//	//std::cout << "[";
//	//for (double k : b)
//	//{
//	//	std::cout << k << "\t,";
//	//}
//	//std::cout << "]\n";
//
//}