#include "hip/hip_runtime.h"
// Example setup for CUDA
#include "CudaAbstraction.cuh"


__global__ void Test(int* a, int* b)
{
	*a += *b;
	// empty kernel
}


int RunCudaTest()
{
	int a = 5, b = 9;

	int* da, * db;

	hipMalloc(&da, sizeof(int));
	hipMalloc(&db, sizeof(int));

	hipMemcpy(da, &a, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(db, &b, sizeof(int), hipMemcpyHostToDevice);

	Test <<<1, 1 >>>(da, db);

	hipMemcpy(&a, da, sizeof(int), hipMemcpyDeviceToHost);
	std::cout << "answer is " << a << std::endl;

	hipFree(da);
	hipFree(db);
	return a;
}
