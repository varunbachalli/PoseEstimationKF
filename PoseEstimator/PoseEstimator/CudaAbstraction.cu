#include "hip/hip_runtime.h"
// Example setup for CUDA
#include "CudaAbstraction.cuh"
#include <stdio.h>
#include<stdlib.h>
#include<time.h>
#include <chrono> 
#include<vector>
#include <hip/device_functions.h>
using namespace std::chrono;


__global__ void Test(int* a, int* b)
{
	*a += *b;
	// empty kernel
}


int RunCudaTest()
{
	int a = 5, b = 9;

	int* da, * db;

	hipMalloc(&da, sizeof(int));
	hipMalloc(&db, sizeof(int));

	hipMemcpy(da, &a, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(db, &b, sizeof(int), hipMemcpyHostToDevice);

	Test <<<1, 1 >>>(da, db);

	hipMemcpy(&a, da, sizeof(int), hipMemcpyDeviceToHost);
	std::cout << "answer is " << a << std::endl;

	hipFree(da);
	hipFree(db);
	return a;
}

void MatrixAddition(int* a, int* b, int* result, int m, int n)
{
	for (int i = 0; i < m; ++i)
	{
		for (int j = 0; j < n; ++j)
		{
			*((result + i*n) + j) = *((a + i * n) + j) + *((b + i * n) + j);
		}
		
	}
	
}

void print(int* arr, int m, int n)
{
	int i, j;
	for (i = 0; i < m; i++)
	{
		for (j = 0; j < n; j++)
		{
			printf("%d ", *((arr + i * n) + j));
		}
		std::cout << std::endl;
	}

}

__global__ void Add_Matrices(int* a, int* b, int* result, int max_threads)
{
	
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < max_threads)
	{
		*(result + id) = *(a + id) + *(b + id);
	}
}

__global__ void Multiply(int* a)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i < 3)
	{
		a[i] = a[i + 1];
		
	}
	
}

int main()
{
	int count = 0;
	for(int l = 0; l < 10000; ++l)
	{	
		int a[] = { 1,2,3,4 };
		int* da;
		hipMalloc(&da, sizeof(a));
		hipMemcpy(da, a, sizeof(a), hipMemcpyHostToDevice);
		Multiply <<<1, 5 >>> (da);
		hipMemcpy(a, da, sizeof(a), hipMemcpyDeviceToHost);
		if (a[0] != 2 && a[1] != 3 && a[2] != 4)
			count++;
		hipFree(da);
	}

	std::cout << count << std::endl;
	//srand(time(0));
	//
	//std::vector<double> time_cpu;
	//std::vector<double> time_gpu;


	//for (int m = 100000; m <= 10000000; m+= 100000)
	//{
	//	/*std::cout << "vector size" << m << std::endl;*/
	//	int* xp = (int*)malloc(m * sizeof(int));
	//	int* yp = (int*)malloc(m * sizeof(int));
	//	int* result = (int*)malloc(m * sizeof(int));
	//	for (int i = 0; i < m; ++i)
	//	{
	//		*xp = rand() % 10000000;
	//		*yp = rand() % 10000000;
	//	}

	//	// time start gpu
	//	auto start = high_resolution_clock::now();
	//	int* da; int* db; int* dres; 

	//	hipMalloc(&da, 4 * m);
	//	hipMalloc(&db, 4 * m );
	//	hipMalloc(&dres, 4 * m);
	//	
	//

	//	hipMemcpy(da, xp, 4 * m, hipMemcpyHostToDevice);
	//	hipMemcpy(db, yp, 4 * m, hipMemcpyHostToDevice);
	//	
	//	Add_Matrices <<<m / 1024 + 1, 1024>>> (da, db, dres, m);

	//	hipMemcpy(result, dres, sizeof(int) * m, hipMemcpyDeviceToHost);
	//	
	//	hipFree(da);
	//	hipFree(db);
	//	hipFree(dres);
	//	// time end gpu
	//	auto stop = high_resolution_clock::now();
	//	double duration = (double)duration_cast<nanoseconds>(stop - start).count();
	//	time_gpu.push_back(duration);
	//	//std::cout << "gpu duration" << duration << std::endl;
	//	start= high_resolution_clock::now();
	//	// time start cpu

	//	for (int i = 0; i < m; ++i)
	//	{
	//		*(result + i) = *(xp + i) + *(yp + i);
	//	}

	//	// time end cpu

	//	free(xp);
	//	free(yp);
	//	free(result);
	//	stop = high_resolution_clock::now();
	//	duration = (double)duration_cast<nanoseconds>(stop - start).count();
	//	time_cpu.push_back(duration);
	//	//std::cout << "cpu duration" << duration << std::endl;
	//	
	//}
	//int gpu_faster = 0;
	//bool starting_first = false;
	//for (int i = 0; i < time_cpu.size(); ++i)
	//{
	//	if (time_gpu[i] - time_cpu[i] < 0.0)
	//	{
	//		gpu_faster++;
	//		if (!starting_first)
	//			std::cout << i << std::endl; starting_first = true;
	//	}

	//}

	//std::cout << "gpu is faster this many times :" << gpu_faster << std::endl;


	//int i = 100;
	//int l = 3;

	//int j = i / l;

	//std::cout << j << std::endl;


	//const int m = 2, n = 2;
	//int xp[m][n] = {{1,2} ,{1,2}};
	//int yp[m][n] = {{1,2} ,{1,2}};
	//int result[m][n];

	//int* da; int* db; int* dres;

	//hipMalloc(&da, sizeof(int) * m * n);
	//hipMalloc(&db, sizeof(int) * m * n);
	//hipMalloc(&dres, sizeof(int) * m * n);

	//hipMemcpy(da, (int*)&xp, sizeof(int) * m * n, hipMemcpyHostToDevice);
	//hipMemcpy(db, (int*)&yp, sizeof(int) * m * n, hipMemcpyHostToDevice);

	//Add_Matrices <<<1, m*n>>> (da, db ,dres);

	//hipMemcpy((int*)&result, dres, sizeof(int) * m * n, hipMemcpyDeviceToHost);
	//print((int*)result, m, n);

	//hipFree(da);
	//hipFree(db);
	//hipFree(dres);
	//std::cout << "Two dimensional Array stuff worked" << std::endl;
	int k;
	std::cin >> k;
	return 0;
}





//
//int main(int argc, char* argv[])
//{
//

//
//
//	MatrixAddition(&xp, &yp, &result, m, n);
//
//	
//
//	MatrixAddition(&xp, &xp);
//	std::cout << sizeof(xp) << std::endl;
//	std::cout << sizeof(int) * 10 * 22 << std::endl;
//	int k;
//	std::cin >> k;
//
//
//
//	return 0;	
//}


// use opengl to plot stuff

